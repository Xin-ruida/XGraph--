#include "hip/hip_runtime.h"
#include "../../shared/timer.hpp"
#include "../../shared/subgraph.cuh"
#include "../../shared/partitioner.cuh"
#include "../../shared/subgraph_generator.cuh"
#include "../../shared/gpu_error_check.cuh"
#include "../../shared/gpu_kernels.cuh"
#include "../../shared/subway_utilities.hpp"
#include "cc_sig.h"

void cc_sig_sync(ArgumentParser arguments)
{
	hipFree(0);

	Timer timer;
	timer.Start();

	GraphStructure graph;
	graph.ReadGraph(arguments.input);

	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime / 1000 << " (s).\n";

	GraphStates<uint> states(graph.num_nodes, true, false);

	for (unsigned int i = 0; i < graph.num_nodes; i++)
	{
		states.value[i] = i;
		states.label1[i] = false;
		states.label2[i] = true;
	}

	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_value, states.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label2, states.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	Subgraph subgraph(graph.num_nodes, graph.num_edges);

	SubgraphGenerator<uint> subgen(graph);

	subgen.generate(graph, states, subgraph);


	Partitioner partitioner;

	timer.Start();

	uint itr = 0;

	while (subgraph.numActiveNodes > 0)
	{
		itr++;

		partitioner.partition(subgraph, subgraph.numActiveNodes);
		// a super iteration
		for (int i = 0; i < partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

			moveUpLabels << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (subgraph.d_activeNodes, states.d_label1, states.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			cc_kernel << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (partitioner.partitionNodeSize[i],
				partitioner.fromNode[i],
				partitioner.fromEdge[i],
				subgraph.d_activeNodes,
				subgraph.d_activeNodesPointer,
				subgraph.d_activeEdgeList,
				graph.d_outDegree,
				states.d_value,
				//d_finished,
				states.d_label1,
				states.d_label2);

			hipDeviceSynchronize();
			gpuErrorcheck(hipPeekAtLastError());
		}

		subgen.generate(graph, states, subgraph);

	}

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime / 1000 << " (s).\n";

	cout << "Number of iterations = " << itr << endl;

	gpuErrorcheck(hipMemcpy(states.value, states.d_value, graph.num_nodes * sizeof(uint), hipMemcpyDeviceToHost));

	utilities::PrintResults(states.value, min(30, graph.num_nodes));

	if (arguments.hasOutput)
		utilities::SaveResults(arguments.output, states.value, graph.num_nodes);
}

