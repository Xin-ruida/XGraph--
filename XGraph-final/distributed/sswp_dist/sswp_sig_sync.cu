#include "hip/hip_runtime.h"
#include "../../shared/timer.hpp"
#include "../../shared/subgraph.cuh"
#include "../../shared/partitioner.cuh"
#include "../../shared/subgraph_generator.cuh"
#include "../../shared/gpu_error_check.cuh"
#include "../../shared/gpu_kernels.cuh"
#include "../../shared/subway_utilities.hpp"
#include "sswp_sig.h"

void sswp_sig_sync(ArgumentParser arguments)
{
	hipFree(0);

	Timer timer;
	timer.Start();

	GraphStructure graph;
	graph.ReadGraph(arguments.input);

	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime / 1000 << " (s).\n";

	GraphStates<uint> states(graph.num_nodes, true, false, graph.num_edges);
	states.ReadEdgeWeight(arguments.input + "w", graph.num_edges);

	for (unsigned int i = 0; i < graph.num_nodes; i++)
	{
		states.value[i] = 0;
		states.label1[i] = false;
		states.label2[i] = false;
	}
	states.value[arguments.sourceNode] = DIST_INFINITY;
	states.label1[arguments.sourceNode] = false;
	states.label2[arguments.sourceNode] = true;


	gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_value, states.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label1, states.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(states.d_label2, states.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	Subgraph subgraph(graph.num_nodes, graph.num_edges, true);

	SubgraphGenerator<uint> subgen(graph);

	subgen.generate(graph, states, subgraph);


	Partitioner partitioner;

	timer.Start();

	uint itr = 0;

	while (subgraph.numActiveNodes > 0)
	{
		itr++;

		partitioner.partition(subgraph, subgraph.numActiveNodes);
		// a super iteration
		for (int i = 0; i < partitioner.numPartitions; i++)
		{
			hipDeviceSynchronize();
			gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdge), hipMemcpyHostToDevice));
			gpuErrorcheck(hipMemcpy(subgraph.d_activeWeightList, subgraph.activeWeightList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(uint), hipMemcpyHostToDevice));
			hipDeviceSynchronize();

			moveUpLabels << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (subgraph.d_activeNodes, states.d_label1, states.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);

			sswp_kernel << < partitioner.partitionNodeSize[i] / 512 + 1, 512 >> > (partitioner.partitionNodeSize[i],
				partitioner.fromNode[i],
				partitioner.fromEdge[i],
				subgraph.d_activeNodes,
				subgraph.d_activeNodesPointer,
				subgraph.d_activeEdgeList,
				subgraph.d_activeWeightList,
				graph.d_outDegree,
				states.d_value,
				//d_finished,
				states.d_label1,
				states.d_label2);

			hipDeviceSynchronize();
			gpuErrorcheck(hipPeekAtLastError());
		}

		subgen.generate(graph, states, subgraph);

	}

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime / 1000 << " (s).\n";

	cout << "Number of iterations = " << itr << endl;

	gpuErrorcheck(hipMemcpy(states.value, states.d_value, graph.num_nodes * sizeof(uint), hipMemcpyDeviceToHost));

	utilities::PrintResults(states.value, min(30, graph.num_nodes));

	if (arguments.hasOutput)
		utilities::SaveResults(arguments.output, states.value, graph.num_nodes);
}

